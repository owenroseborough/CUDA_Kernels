// matrix mul one thread one col.cu
// Owen Roseborough

#include <iostream>
#include <string>
#include <cassert>
#include <ctime>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>

using namespace std;

#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr, "CUDA Error: %s in %s at line %d\n",
                hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#define cudaCheckKernel() { gpuKernelCheck(__FILE__, __LINE__); } // Use this after kernel launches to check for errors
inline void gpuKernelCheck(const char *file, int line)
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA Kernel Launch Error: %s at %s:%d\n",
                hipGetErrorString(err), file, line);
        exit(err);
    }

    err = hipDeviceSynchronize(); // Optional: sync to catch async errors
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA Sync Error: %s at %s:%d\n",
                hipGetErrorString(err), file, line);
        exit(err);
    }
}

__global__ void matrixMulKernel(float *A_d, float *B_d, float *C_d, int N);

int main(void)
{
   int N = 1024; 
   float *A_h = (float*)malloc(N * N * sizeof(float));
   float *B_h = (float*)malloc(N * N * sizeof(float));
   float *C_h = (float*)malloc(N * N * sizeof(float));
   if (A_h == NULL || B_h == NULL || C_h == NULL){
      cout << "error allocating host memory";
      return 0;
   }
   for (int i = 0; i < N * N; i++) {
      A_h[i] = 1.0f;
      B_h[i] = 2.0f;
   }
     
   float *A_d; float *B_d; float *C_d; // pointers to device memory

   // allocate matrix arrays on device
   cudaCheckError(hipMalloc((void **) &A_d, sizeof(float) * N * N));
   cudaCheckError(hipMalloc((void **) &B_d, sizeof(float) * N * N));
   cudaCheckError(hipMalloc((void **) &C_d, sizeof(float) * N * N));
   
   // transfering A_h and B_h to device
   cudaCheckError(hipMemcpy(A_d, A_h, sizeof(float) * N * N, hipMemcpyHostToDevice));
   cudaCheckError(hipMemcpy(B_d, B_h, sizeof(float) * N * N, hipMemcpyHostToDevice));
   
   //call matrix mul kernel
   dim3 blockSize(32);
   dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
   matrixMulKernel<<<gridSize, blockSize>>>(A_d, B_d, C_d, N);

   cudaCheckKernel();

   // retrieve out matrix from device: C_d to C_h
   cudaCheckError(hipMemcpy(C_h, C_d, sizeof(float) * N * N, hipMemcpyDeviceToHost));

   // print resultant matrix
   for(int row = 0; row < N; row++){
      for(int col = 0; col < N; col++){
         cout << C_h[row * N + col] << " ";
      }
      cout << endl;
   }

   // cleanup
   free(A_h); free(B_h); free(C_h);
   hipFree(A_d); hipFree(B_d); hipFree(C_d);
}

__global__  
void matrixMulKernel(float *A_d, float *B_d, float *C_d, int N){
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   if(index < N){
      //one thread produces a col in C_d
      for(int row = 0; row < N; row++){
         //iterate over row and col to create output
         float Pvalue = 0;
         for(int k = 0; k < N; k++){
            Pvalue += A_d[row*N + k] * B_d[k*N + index];
         }
         C_d[row * N + index] = Pvalue;
      }
   }
}
