#include "hip/hip_runtime.h"
// segmented parallel scan.cu
// Owen Roseborough

#include <iostream>
#include <string>
#include <cassert>
#include <ctime>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>
#include "cuda macros.h"

using namespace std;

__global__ void firstKernel(unsigned int *input_d, unsigned int *output_d, int maxThreadsX);

#define INPUTSIZE 1000000

int main(void)
{
    unsigned int *input_h;
    input_h = (unsigned int *)mallocCheckError(malloc(sizeof(unsigned int) * INPUTSIZE));

    // randomly initialize input_h
    srand(time(NULL));
    for(unsigned int i = 0; i < INPUTSIZE; i++){
        input_h[i] = (rand() % 5) + 1;  // rand() % 5 gives 0–4, +1 makes it 1–5
    }

    unsigned int *output_h;
    output_h = (unsigned int *)mallocCheckError(malloc(sizeof(unsigned int) * INPUTSIZE));
    
    // determine size of kernel blocks and grid
    int devCount;
    cudaCheckError(hipGetDeviceCount(&devCount));
    hipDeviceProp_t devProp;
    int maxThreadsX = 0;
    int device = 0;
    for(unsigned int i = 0; i < devCount; i++){
        cudaCheckError(hipGetDeviceProperties(&devProp, i));
        if(devProp.maxThreadsDim[0] > maxThreadsX){
            maxThreadsX = devProp.maxThreadsDim[0];
            device = i;
        }
    }
    cudaCheckError(hipSetDevice(device)); //set device to one with max thread capability in x axis
    
    // allocate input & output arrays on device
    unsigned int *input_d, *output_d;     // pointers to device memory
    cudaCheckError(hipMalloc((void **) &input_d, sizeof(unsigned int) * INPUTSIZE));
    cudaCheckError(hipMalloc((void **) &output_d, sizeof(unsigned int) * INPUTSIZE));
    
    // send input_h from host to device
    cudaCheckError(hipMemcpy(input_d, input_h, sizeof(unsigned int) * INPUTSIZE, hipMemcpyHostToDevice));

    dim3 blockSize(maxThreadsX, 1);
    dim3 gridSize(ceil(INPUTSIZE / maxThreadsX),1);

    //call segmented parallel scan kernels
    firstKernel<<<gridSize, blockSize, 2*maxThreadsX*sizeof(unsigned int)>>>(input_d, output_d, maxThreadsX);

    cudaCheckKernel();

    // retrieve output_d from device
    cudaCheckError(hipMemcpy(output_h, output_d, sizeof(unsigned int) * INPUTSIZE, hipMemcpyDeviceToHost));

    // compute scan sequentially on host
    unsigned int *test_output_h;
    test_output_h = (unsigned int *)mallocCheckError(malloc(sizeof(unsigned int) * INPUTSIZE));
    test_output_h[0] = input_h[0];
    for(unsigned int i = 1; i < INPUTSIZE; i++){
        test_output_h[i] = test_output_h[i - 1] + input_h[i];
    }
    // compare sequential result to parallel to ensure they are the same
    for(unsigned int i = 0; i < INPUTSIZE; i++){
        assert(test_output_h[i] == output_h[i]);
    }

    // cleanup
    free(input_h); free(output_h); free(test_output_h);
    hipFree(input_d); hipFree(output_d);
}

__global__ void firstKernel(unsigned char *input_d, unsigned char *output_d, int maxThreadsX){

    extern __shared__ unsigned int buffer[];
    unsigned int *firstBuf = buffer;
    unsigned int *secondBuf = &firstBuf[maxThreadsX];

    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < INPUTSIZE){
        firstBuf[threadIdx.x] = input_d[i];
    } else{
        firstBuf[threadIdx.x] = 0.0f;
    }
    unsigned int counter = 0;
    for(unsigned int stride = 1; stride < blockDim.x; stride *= 2){
        __syncthreads();
        if(counter % 2 == 0 && threadIdx.x >= stride)
            secondBuf[threadIdx.x] = firstBuf[threadIdx.x] + firstBuf[threadIdx.x - stride];
        else if(threadIdx.x >= stride)
            firstBuf[threadIdx.x] = secondBuf[threadIdx.x] + secondBuf[threadIdx.x - stride];
        counter += 1;
    }
    if(i < INPUTSIZE){
        if(counter % 2 == 0){ 
            output_d[i] = firstBuf[threadIdx.x];
        } else {
            output_d[i] = secondBuf[threadIdx.x];
        }
    }
}
